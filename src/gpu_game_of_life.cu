#include "hip/hip_runtime.h"
#include "gpu_game_of_life.hpp"
#include <cassert>
#include <hip/hip_runtime.h>
#include "time.hpp"

#define CHECK_CUDA_CALL(ret) check_cuda_call_impl(ret, __FILE__, __LINE__)
#define CHECK_LAST_CUDA_CALL() check_cuda_call_impl(hipGetLastError(), __FILE__, __LINE__)

void check_cuda_call_impl(const hipError_t err, const char* fileName, const int lineNumber) {
	if (err != hipSuccess) {
		fprintf(stderr, "CUDA error at %s:%d: %s\n", fileName, lineNumber, hipGetErrorString(err));
		exit(1);
	}
}

GPU_Board::GPU_Board(uint32_t width, uint32_t height)
	: Board(width, height) {
	// TODO: Can I use pitched ptrs? Apparently it makes sure the memory allocated is accessed in the most efficient way possible.
	CHECK_CUDA_CALL(hipHostMalloc(&m_HostCells, width * height * sizeof(uint8_t)));
	CHECK_CUDA_CALL(hipMalloc(&m_DeviceCells, width * height * sizeof(uint8_t)));
}

GPU_Board::GPU_Board(GPU_Board&& other)
	: Board(0, 0) {
	*this = std::move(other);
}

GPU_Board::~GPU_Board() {
	CHECK_CUDA_CALL(hipHostFree(m_HostCells));
	CHECK_CUDA_CALL(hipFree(m_DeviceCells));
}

GPU_Board& GPU_Board::operator=(GPU_Board&& other) {
	if (this != &other) {
		CHECK_CUDA_CALL(hipHostFree(m_HostCells));
		CHECK_CUDA_CALL(hipFree(m_DeviceCells));

		m_Width = other.m_Width;
		m_Height = other.m_Height;
		m_HostCells = other.m_HostCells;
		m_DeviceCells = other.m_DeviceCells;

		other.m_HostCells = nullptr;
		other.m_DeviceCells = nullptr;
	}

	return *this;
}

void GPU_Board::set_cell(uint32_t x, uint32_t y, const Cell cell) {
	assert(x < m_Width && y < m_Height);

	m_HostCells[y * m_Width + x] = cell;
}

Cell GPU_Board::get_cell(uint32_t x, uint32_t y) const {
	assert(x < m_Width && y < m_Height);

	return m_HostCells[y * m_Width + x];
}

Cell GPU_Board::get_cell_or_dead(uint32_t x, uint32_t y) const {
	if (x < m_Width && y < m_Height) {
		return m_HostCells[y * m_Width + x];
	}

	return Cell::DEAD;
}

#define BRANCHLESS true

// TODO: Consolidate input and output boards into one array
__global__ void game_of_life_kernel(uint8_t* input, uint8_t* output, uint32_t boardWidth, uint32_t boardHeight) {
	int32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	int32_t y = blockIdx.y * blockDim.y + threadIdx.y;

	int neighbors = 0;

	if (y > 0) {
		neighbors += input[(y - 1) * boardWidth + x];

		if (x > 0) {
			neighbors += input[(y - 1) * boardWidth + x - 1];
		}

		if (x < boardWidth - 1) {
			neighbors += input[(y - 1) * boardWidth + x + 1];
		}
	}

	if (x > 0) {
		neighbors += input[y * boardWidth + x - 1];
	}

	if (x < boardWidth - 1) {
		neighbors += input[y * boardWidth + x + 1];
	}

	if (y < boardHeight - 1) {
		neighbors += input[(y + 1) * boardWidth + x];

		if (x > 0) {
			neighbors += input[(y + 1) * boardWidth + x - 1];
		}

		if (x < boardWidth - 1) {
			neighbors += input[(y + 1) * boardWidth + x + 1];
		}
	}

	Cell current = (Cell)input[y * boardWidth + x];

#if BRANCHLESS == true
	output[y * boardWidth + x] = (int)current * (neighbors == 2 || neighbors == 3) + (1 - (int)current) * (neighbors == 3);
#else
	if (current == Cell::ALIVE) {
		if (neighbors == 2 || neighbors == 3) {
			output[y * boardWidth + x] = Cell::ALIVE;
		} else {
			output[y * boardWidth + x] = Cell::DEAD;
		}
	} else {
		if (neighbors == 3) {
			output[y * boardWidth + x] = Cell::ALIVE;
		} else {
			output[y * boardWidth + x] = Cell::DEAD;
		}
	}
#endif
}

#define BANDWIDTH_MEASUREMENT false

GPU_Board& GPU_GameOfLife::step() {
	constexpr int BLOCK_SIZE = 32;
	dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);

	uint32_t width = m_CurrentBoard.width();
	uint32_t height = m_CurrentBoard.height();

	dim3 blockCount((int)ceil((float)width / BLOCK_SIZE), (int)ceil((float)height / BLOCK_SIZE));

#if BANDWIDTH_MEASUREMENT == true
	auto start = get_time_secs();
#endif
	CHECK_CUDA_CALL(hipMemcpy(m_CurrentBoard.device_cells(), m_CurrentBoard.host_cells(), width * height * sizeof(uint8_t), hipMemcpyHostToDevice));
#if BANDWIDTH_MEASUREMENT == true
	auto end = get_time_secs();
	auto time = end - start;
	printf("H2D time: %f sec, bandwidth: %f GB/s\n", time, (width * height * sizeof(uint8_t) / 1e9) / time);
#endif

	game_of_life_kernel<<<blockCount, blockDim>>>((uint8_t*)m_CurrentBoard.device_cells(), (uint8_t*)m_NextBoard.device_cells(), width, height);
	CHECK_LAST_CUDA_CALL();

	CHECK_CUDA_CALL(hipDeviceSynchronize());

#if BANDWIDTH_MEASUREMENT == true
	start = get_time_secs();
#endif
	CHECK_CUDA_CALL(hipMemcpy(m_NextBoard.host_cells(), m_NextBoard.device_cells(), width * height * sizeof(uint8_t), hipMemcpyDeviceToHost));
#if BANDWIDTH_MEASUREMENT == true
	end = get_time_secs();
	time = end - start;
	printf("D2H time: %f sec, bandwidth: %f GB/s\n", time, (width * height * sizeof(uint8_t) / 1e9) / time);
#endif

	std::swap(m_CurrentBoard, m_NextBoard);

	return m_CurrentBoard;
}

std::string get_device_name() {
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	return std::string(prop.name);
}